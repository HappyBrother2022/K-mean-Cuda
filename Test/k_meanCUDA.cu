#include "hip/hip_runtime.h"
﻿#include "k_meanCUDA.cuh"
#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "math.h"
#include ""
#include "k_mean.h"

#define tile 32
//calculate the Distance between two points : formula d=√ ∑i(x−x2i)^2+(y−yi)^2 - 
//Where (x,y) represent the centroid’s coordinates, and (xi,yi) represent the data-point’s coordinates.
//Device to execute in device - meaning call the function in device 

__device__ double calculatedistanceGPU(unit* point1, unit* point2) {
	return (double)sqrt((double)pow(point1->dim1 - point2->dim1, 2) + (double)pow(point1->dim2 - point2->dim2, 2) + (double)pow(point1->dim3 - point2->dim3, 2) + (double)pow(point1->dim4 - point2->dim4, 2));
}
//static and funtion pointers not allowed 
// We find the closest centroid to the points
__global__ void closestcentroidGPU(unit* points, unit* centroids, int numofcentr, int numofpoints) {

	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	if (tid < numofpoints) {
		double dist = 0;
		double firstdistance = calculatedistanceGPU(&points[tid], &centroids[0]);
		points[tid].cluster = 0;
		for (int i = 1; i < numofcentr; i++) {
			dist = calculatedistanceGPU(&points[tid], &centroids[i]);// calculating the distance between the centroid and the point
			if (dist <= firstdistance) //getting the smaller distance till we get the smallest(end of forloop) distance
			{ 
				points[tid].cluster = i;
				firstdistance = dist;
			}
		}
	}
}


//
__global__ void closestcentroidSharedGPU(unit* points, unit* centroids, int numofcentr, int numofpoints) {

	int threadsPerBlock = blockDim.x * blockDim.y * blockDim.z;
	int threadPosInBlock = threadIdx.x + blockDim.x * threadIdx.y + blockDim.x * blockDim.y * threadIdx.z;
	int blockPosInGrid = blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
	int tid = blockPosInGrid * threadsPerBlock + threadPosInBlock;

	//Managing comunication and syncronization
	//within a block, threads share data via shared memory

	__shared__ unit sh_points[tile*tile];
	__shared__ unit sh_centrs[4];
	
	// finding which point belongs to which centroid

	if (tid < numofpoints) {
		sh_points[threadPosInBlock] = points[tid];
		if (tid%threadsPerBlock ==0) {
			for (int i = 0; i < numofcentr; i++) {
				sh_centrs[i] = centroids[i];
			}
			
		}
	//Synchronizes all threads within a block 
	// prevent data hazards
		__syncthreads();

		double dist = 0;
		double firstdistance = calculatedistanceGPU(&sh_points[threadPosInBlock], &sh_centrs[0]);
		sh_points[threadPosInBlock].cluster = 0;
	
	//calculate smallest distance between a point and centeroid ina particular block

		for (int i = 1; i < numofcentr; i++) {
			dist = calculatedistanceGPU(&sh_points[threadPosInBlock], &sh_centrs[i]);
			if (dist <= firstdistance) {
				sh_points[threadPosInBlock].cluster = i;
				firstdistance = dist;
			}
		}
		__syncthreads();

		// Assinging all the shared points in a block to the particular thread id

		points[tid] = sh_points[threadPosInBlock];
		
		__syncthreads();
	}
}